
#include <hip/hip_runtime.h>
/*
 * Original project released under the public domain by Ville Timonen in 2013
 *
 * All changes and improvements Copyright (c) 2013-2016 by Microway, Inc.
 *
 * This file is part of Microway gpu-burn
 *
 * Microway gpu-burn is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Microway gpu-burn is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with gpu-burn.  If not, see <http://www.gnu.org/licenses/>
 */

// Actually, there are no rounding errors due to results being accumulated in an arbitrary order..
// Therefore EPSILON = 0.0f is OK
#define EPSILON 0.001f
#define EPSILOND 0.0000001

extern "C" __global__ void compare(float *C, int *faultyElems, size_t iters) {
	size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		gridDim.x*blockDim.x + // W
		blockIdx.x*blockDim.x + threadIdx.x; // X

	int myFaulty = 0;
	for (size_t i = 1; i < iters; ++i)
		if (fabsf(C[myIndex] - C[myIndex + i*iterStep]) > EPSILON)
			myFaulty++;

	atomicAdd(faultyElems, myFaulty);
}

extern "C" __global__ void compareD(double *C, int *faultyElems, size_t iters) {
	size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		gridDim.x*blockDim.x + // W
		blockIdx.x*blockDim.x + threadIdx.x; // X

	int myFaulty = 0;
	for (size_t i = 1; i < iters; ++i)
		if (fabs(C[myIndex] - C[myIndex + i*iterStep]) > EPSILOND)
			myFaulty++;

	atomicAdd(faultyElems, myFaulty);
}
